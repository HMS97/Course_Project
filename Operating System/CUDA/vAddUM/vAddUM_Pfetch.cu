#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void vecAdd(int n, float *a, float *b, float *c)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i+=stride)
    c[i] = a[i] + b[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y, *z;

  float msec;
  hipEvent_t start, stop;

  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  hipMallocManaged(&z, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Prefetch the data to the GPU
  int device;
  hipGetDevice(&device);
  hipMemPrefetchAsync(x, N*sizeof(float), device, NULL);
  hipMemPrefetchAsync(y, N*sizeof(float), device, NULL);



  int blockSize = 256;
  int numBlocks = 12; // good enough for P620 
  hipEventRecord(start);
  vecAdd<<<numBlocks, blockSize>>>(N, x, y, z);
  hipEventRecord(stop);
  // Wait for GPU to finish before accessing on host
  hipEventSynchronize(stop);  

  hipEventElapsedTime(&msec, start, stop);
  printf("Kernel time: %f ms\n", msec);
  
  hipMemPrefetchAsync(z, N*sizeof(float), hipCpuDeviceId, NULL);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(z[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  hipFree(z);
  
  return 0;
}
